//          Copyright Carl Pearson 2020 - 2021.
// Distributed under the Boost Software License, Version 1.0.
//    (See accompanying file LICENSE or copy at
//    https://www.boost.org/LICENSE_1_0.txt)

#include "../include/env.hpp"
#include "../support/type.hpp"

#include <mpi.h>
#include <nvToolsExt.h>

#include <iostream>

int main(int argc, char **argv) {
  MPI_Init(&argc, &argv);

  int rank = 0, size = 1;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Comm_size(MPI_COMM_WORLD, &size);
  if (1 != size) {
    std::cerr << "ERROR: requires exactly 1 rank\n";
    exit(1);
  }

  std::cerr << "TEST: contiguous\n";
  int n = 800;

  MPI_Datatype ty = make_contiguous_contiguous(n);
  MPI_Type_commit(&ty);

  char *hostSend = new char[n];
  char *hostRecv = new char[n];
  char *deviceSend, *deviceRecv;
  hipMalloc(&deviceSend, sizeof(char) * n);
  hipMalloc(&deviceRecv, sizeof(char) * n);
  MPI_Request reqSend, reqRecv;

  // host send / recv
  std::cerr << "HOST\n";
  MPI_Isend(hostSend, 1, ty, rank, 0, MPI_COMM_WORLD, &reqSend);
  MPI_Irecv(hostRecv, 1, ty, rank, 0, MPI_COMM_WORLD, &reqRecv);
  MPI_Wait(&reqSend, MPI_STATUS_IGNORE);
  MPI_Wait(&reqRecv, MPI_STATUS_IGNORE);

  // device send/recv
  std::cerr << "DEVICE 1\n";
  nvtxRangePush("device");
  MPI_Isend(deviceSend, 1, ty, rank, 0, MPI_COMM_WORLD, &reqSend);
  MPI_Irecv(deviceRecv, 1, ty, rank, 0, MPI_COMM_WORLD, &reqRecv);
  MPI_Wait(&reqSend, MPI_STATUS_IGNORE);
  MPI_Wait(&reqRecv, MPI_STATUS_IGNORE);
  nvtxRangePop();

  // device send/recv
  std::cerr << "DEVICE 2\n";
  nvtxRangePush("device");
  MPI_Isend(deviceSend, 1, ty, rank, 0, MPI_COMM_WORLD, &reqSend);
  MPI_Irecv(deviceRecv, 1, ty, rank, 0, MPI_COMM_WORLD, &reqRecv);
  MPI_Wait(&reqSend, MPI_STATUS_IGNORE);
  MPI_Wait(&reqRecv, MPI_STATUS_IGNORE);
  nvtxRangePop();

#if 0
  environment::noTempi = true;

  nvtxRangePush("noTempi");
  // device send/recv
  MPI_Isend(deviceSend, 1, ty, rank, 0, MPI_COMM_WORLD, &reqSend);
  MPI_Irecv(deviceRecv, 1, ty, rank, 0, MPI_COMM_WORLD, &reqRecv);
  MPI_Wait(&reqSend, MPI_STATUS_IGNORE);
  MPI_Wait(&reqRecv, MPI_STATUS_IGNORE);
  nvtxRangePop();

  environment::noTempi = false;
#endif
  MPI_Finalize();

  delete[] hostSend;
  delete[] hostRecv;
  hipFree(deviceSend);
  hipFree(deviceRecv);

  return 0;
}