//          Copyright Carl Pearson 2020 - 2021.
// Distributed under the Boost Software License, Version 1.0.
//    (See accompanying file LICENSE or copy at
//    https://www.boost.org/LICENSE_1_0.txt)

#include <mpi.h>
#include <nvToolsExt.h>

#include "../include/env.hpp"

int main(int argc, char **argv) {
  MPI_Init(&argc, &argv);

  int rank;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);

  float *hostSend = new float[100];
  float *hostRecv = new float[100];
  float *deviceSend, *deviceRecv;
  hipMalloc(&deviceSend, sizeof(float) * 100);
  hipMalloc(&deviceRecv, sizeof(float) * 100);
  MPI_Request reqSend, reqRecv;

  // host send / recv
  MPI_Isend(hostSend, 100, MPI_FLOAT, rank, 0, MPI_COMM_WORLD, &reqSend);
  MPI_Irecv(hostRecv, 100, MPI_FLOAT, rank, 0, MPI_COMM_WORLD, &reqRecv);
  MPI_Wait(&reqSend, MPI_STATUS_IGNORE);
  MPI_Wait(&reqRecv, MPI_STATUS_IGNORE);

  // device send/recv
  nvtxRangePush("TEMPI");
  MPI_Isend(deviceSend, 100, MPI_FLOAT, rank, 0, MPI_COMM_WORLD, &reqSend);
  MPI_Irecv(deviceRecv, 100, MPI_FLOAT, rank, 0, MPI_COMM_WORLD, &reqRecv);
  MPI_Wait(&reqSend, MPI_STATUS_IGNORE);
  MPI_Wait(&reqRecv, MPI_STATUS_IGNORE);
  nvtxRangePop();

  environment::noTempi = true;

  nvtxRangePush("noTempi");
  // device send/recv
  MPI_Isend(deviceSend, 100, MPI_FLOAT, rank, 0, MPI_COMM_WORLD, &reqSend);
  MPI_Irecv(deviceRecv, 100, MPI_FLOAT, rank, 0, MPI_COMM_WORLD, &reqRecv);
  MPI_Wait(&reqSend, MPI_STATUS_IGNORE);
  MPI_Wait(&reqRecv, MPI_STATUS_IGNORE);
  nvtxRangePop();

  environment::noTempi = false;
  MPI_Finalize();

  delete[] hostSend;
  delete[] hostRecv;
  hipFree(deviceSend);
  hipFree(deviceRecv);

  return 0;
}