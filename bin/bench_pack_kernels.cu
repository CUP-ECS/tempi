//          Copyright Carl Pearson 2020 - 2021.
// Distributed under the Boost Software License, Version 1.0.
//    (See accompanying file LICENSE or copy at
//    https://www.boost.org/LICENSE_1_0.txt)

#include "statistics.hpp"

#include "../include/allocators.hpp"
#include "../include/cuda_runtime.hpp"
#include "../include/pack_kernels.cuh"

#include <nvToolsExt.h>

#include <chrono>
#include <cstring> //memset
#include <iostream>
#include <sstream>

typedef std::chrono::system_clock Clock;
typedef std::chrono::duration<double> Duration;
typedef std::chrono::time_point<Clock, Duration> Time;

struct BenchResult {
  int64_t size;
  double packTime;
  double unpackTime;
};

struct BenchArgs {
  int64_t numBlocks;
  int64_t blockLength;
  int64_t stride;
  int64_t count; // number of objects
};

BenchResult bench(const BenchArgs &args, // message datatype
                  const int nIters,      // iterations to measure
                  const bool stage,      // pack into host / unpack from host
                  const char *name = "<unnamed>") {

  int64_t objExt = (args.numBlocks - 1) * args.stride + args.blockLength;
  const int64_t packedSize = args.numBlocks * args.blockLength;

  char *src{}, *dst{};
  CUDA_RUNTIME(hipSetDevice(0));
  CUDA_RUNTIME(hipMalloc(&src, objExt * args.count));
  if (stage) {
    dst = hostAllocator.allocate(packedSize * args.count);
  } else {
    CUDA_RUNTIME(hipMalloc(&dst, packedSize * args.count));
  }

  if (stage) {
    CUDA_RUNTIME(hipMemset(src, 0xFE, objExt * args.count));
    std::memset(dst, 0x00, packedSize * args.count);
  }

  hipStream_t stream;
  hipEvent_t start, stop;
  CUDA_RUNTIME(hipStreamCreate(&stream));
  CUDA_RUNTIME(hipEventCreate(&start));
  CUDA_RUNTIME(hipEventCreate(&stop));

  Pack2DConfig config(args.blockLength, args.numBlocks);

  dim3 gd = config.dim_grid(args.count);
  dim3 bd = config.dim_block();

  // dimBlock = 32;
  // dimGrid = 1;

  std::cerr << " [" << gd.x << " " << gd.y << " " << gd.z << "]x[" << bd.x
            << " " << bd.y << " " << bd.z << "] ";

#if 0
  std::cerr << "[" << dimGrid.x << " " << dimGrid.y << " " << dimGrid.z <<
  std::endl;
#endif

  Statistics packStats;
  nvtxRangePush(name);
  for (int n = 0; n < nIters; ++n) {

    CUDA_RUNTIME(hipEventRecord(start, stream));
    config.packfn<<<gd, bd, 0, stream>>>(dst, src, args.count, args.blockLength,
                                         args.numBlocks, args.stride, objExt);
    CUDA_RUNTIME(hipEventRecord(stop, stream));
    CUDA_RUNTIME(hipEventSynchronize(stop));
    CUDA_RUNTIME(hipGetLastError());
    float millis;
    CUDA_RUNTIME(hipEventElapsedTime(&millis, start, stop));
    packStats.insert(millis / 1024.0);
  }
  nvtxRangePop();

  if (stage) {
    for (size_t i = 0; i < packedSize * args.count; ++i) {
      if (dst[i] != char(0xFE)) {
        exit(-1);
      }
    }
  }

  CUDA_RUNTIME(hipFree(src));
  if (stage) {
    hostAllocator.deallocate(dst, packedSize * args.count);
  } else {
    CUDA_RUNTIME(hipFree(dst));
  }

  CUDA_RUNTIME(hipEventDestroy(start));
  CUDA_RUNTIME(hipEventDestroy(stop));
  CUDA_RUNTIME(hipStreamDestroy(stream));

  return BenchResult{.size = packedSize * args.count,
                     .packTime = packStats.trimean(),
                     .unpackTime = 0};
}

int main(int argc, char **argv) {

  int nIters = 30;

  std::vector<bool> stages{
      false, true}; // whether to one-shot pack device-host / unpack host-device

  BenchResult result;

  /* 2D packing
   */

  std::vector<int> targets{1024, 1024 * 1024};
  // targets = {1024 * 1024};

  std::vector<int> counts{1, 2};
  // counts = {1};

  std::cout << "s,one-shot,count,obj size(B),stride,blocklengths,s,MiB/s";
  std::cout << std::endl << std::flush;

  std::vector<int> blockLengths{1,  2,  4,  6,  8,  12,  16, 20,
                                24, 28, 32, 64, 96, 128, 256};
  // blockLengths = {1};
  std::vector<int> strides{16, 256};
  // strides = {16};

  for (bool stage : stages) {
    for (int target : targets) {
      for (int count : counts) {
        for (int stride : strides) {
          for (int blockLength : blockLengths) {

            int numBlocks = target / blockLength;

            if (numBlocks > 0 && stride >= blockLength) {

              std::string s;
              s += std::to_string(stage);
              s += "|" + std::to_string(count);
              s += "|" + std::to_string(target);
              s += "|" + std::to_string(stride);
              s += "|" + std::to_string(blockLength);

              std::cout << s;
              std::cout << "," << stage;
              std::cout << "," << count;
              std::cout << "," << target;
              std::cout << "," << stride;
              std::cout << "," << blockLength;
              std::cout << std::flush;

              BenchArgs args{.numBlocks = numBlocks,
                             .blockLength = blockLength,
                             .stride = stride,
                             .count = count};

              result = bench(args, nIters, stage, s.c_str());
              std::cout << "," << result.packTime;
              std::cout << ","
                        << double(result.size) / 1024.0 / 1024.0 /
                               result.packTime;
              std::cout << std::flush;
              std::cout << std::endl << std::flush;
            }
          }
        }
      }
    }
  }

  return 0;
}
