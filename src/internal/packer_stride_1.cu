#include "hip/hip_runtime.h"
#include "packer_stride_1.hpp"

#include "cuda_runtime.hpp"
#include "dim3.hpp"
#include "logging.hpp"

/* pack blocks of bytes separated a stride

    the z dimension is used for the incount

each thread loads N bytes of a block
 */
template <unsigned N>
__global__ static void
pack_bytes(void *__restrict__ outbuf,
           int position, // location in the output buffer to start packing (B)
           const void *__restrict__ inbuf,
           const int incount,    // number of datatypes to pack
           unsigned blockLength, // block length (B)
           unsigned count,       // count of blocks in a group
           unsigned stride       // stride (B) between start of blocks in group
) {

  assert(blockLength % N == 0); // N should evenly divide block length
  assert(count >= 1);

  // as the input space may be large, incount * extent may be over 2G
  const uint64_t extent = (count - 1) * stride + blockLength;

  const unsigned int tz = blockDim.z * blockIdx.z + threadIdx.z;
  const unsigned int ty = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned int tx = blockDim.x * blockIdx.x + threadIdx.x;

  char *__restrict__ op = reinterpret_cast<char *>(outbuf) + position;
  const char *__restrict__ ip = reinterpret_cast<const char *>(inbuf);

  for (int z = tz; z < incount; z += gridDim.z * blockDim.z) {
    // each packed datatype will take count * blockLength bytes in outbuf
    char *__restrict__ dst = op + z * blockLength * count;
    // each datatype input has extent separating their starts
    const char *__restrict__ src = ip + z * extent;

    if (tz == 0 && ty == 0 && tx == 0) {
      printf("src offset =%d\n", z * extent);
    }

    // x direction handle the blocks, y handles the block counts
    for (unsigned y = ty; y < count; y += gridDim.y * blockDim.y) {
      for (unsigned x = tx; x < blockLength / N; x += gridDim.x * blockDim.x) {
        unsigned bo = y * blockLength + x * N;
        unsigned bi = y * stride + x * N;
        // printf("%u -> %u\n", bi, bo);

#if 0
        {
          uintptr_t ioff = uintptr_t(src + bi) - uintptr_t(inbuf);
          uintptr_t ooff = uintptr_t(dst + bo) - uintptr_t(outbuf);
          if (ioff >= 4294963208ull) {
            printf("ioff=%lu bi=%u, z=%d, z*ext=%lu\n", ioff, bi, z, z*extent);
          }
          if (ooff >= 8388608) {
            printf("ooff=%lu bo=%u, z=%d, z*bl*cnt=%d\n", ooff, bo, z, z * blockLength * count);
          }
        }
#endif

        if (N == 1) {
          dst[bo] = src[bi];
        } else if (N == 2) {
          uint16_t *__restrict__ d = reinterpret_cast<uint16_t *>(dst + bo);
          const uint16_t *__restrict__ s =
              reinterpret_cast<const uint16_t *>(src + bi);
          *d = *s;
        } else if (N == 4) {
          uint32_t *__restrict__ d = reinterpret_cast<uint32_t *>(dst + bo);
          const uint32_t *__restrict__ s =
              reinterpret_cast<const uint32_t *>(src + bi);
          *d = *s;
        } else if (N == 8) {
          uint64_t *__restrict__ d = reinterpret_cast<uint64_t *>(dst + bo);
          const uint64_t *__restrict__ s =
              reinterpret_cast<const uint64_t *>(src + bi);
          *d = *s;
        }
      }
    }
  }
}

/* unpack

    griddim.z is the count

each thread loads N bytes of a block
*/
template <unsigned N>
__global__ static void unpack_bytes(
    void *__restrict__ outbuf,
    int position, // location in the output buffer to start unpacking (B)
    const void *__restrict__ inbuf,
    const int outcount,   // number of datatypes to unpack
    unsigned blockLength, // block length (B)
    unsigned count,       // count of blocks in a group
    unsigned stride       // stride (B) between start of blocks in group
) {

  assert(blockLength % N == 0); // N should evenly divide block length

  const uint64_t extent = (count - 1) * stride + blockLength;

  const unsigned int tz = blockDim.z * blockIdx.z + threadIdx.z;
  const unsigned int ty = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned int tx = blockDim.x * blockIdx.x + threadIdx.x;

  char *__restrict__ op = reinterpret_cast<char *>(outbuf) + position;
  const char *__restrict__ ip = reinterpret_cast<const char *>(inbuf);

  for (int z = tz; z < outcount; z += gridDim.z * blockDim.z) {
    // each datatype will have stride * count separating their starts in outbuf
    // each packed datatype has blockLength * count separating their starts
    char *__restrict__ dst = op + z * extent;
    const char *__restrict__ src = ip + z * blockLength * count;

    // x direction handle the blocks, y handles the block counts
    for (unsigned y = ty; y < count; y += gridDim.y * blockDim.y) {
      for (unsigned x = tx; x < blockLength / N; x += gridDim.x * blockDim.x) {
        unsigned bi = y * blockLength + x * N;
        unsigned bo = y * stride + x * N;
        // printf("%u -> %u\n", bi, bo);

        if (N == 1) {
          dst[bo] = src[bi];
        } else if (N == 2) {
          uint16_t *__restrict__ d = reinterpret_cast<uint16_t *>(dst + bo);
          const uint16_t *__restrict__ s =
              reinterpret_cast<const uint16_t *>(src + bi);
          *d = *s;
        } else if (N == 4) {
          uint32_t *__restrict__ d = reinterpret_cast<uint32_t *>(dst + bo);
          const uint32_t *__restrict__ s =
              reinterpret_cast<const uint32_t *>(src + bi);
          *d = *s;
        } else if (N == 8) {
          uint64_t *__restrict__ d = reinterpret_cast<uint64_t *>(dst + bo);
          const uint64_t *__restrict__ s =
              reinterpret_cast<const uint64_t *>(src + bi);
          *d = *s;
        }
      }
    }
  }
}

PackerStride1::PackerStride1(unsigned off, unsigned blockLength, unsigned count,
                             unsigned stride) {
  offset_ = off;
  blockLength_ = blockLength;
  assert(blockLength_ > 0);
  count_ = count;
  stride_ = stride;

  // blocklength is a multiple of wordsize
  // offset is a multiple of wordsize
  // wordsize is at most 8
  wordSize_ = 1;
  while (0 == blockLength % (wordSize_ * 2) && 0 == offset_ % (wordSize_ * 2) &&
         (wordSize_ * 2 <= 8)) {
    wordSize_ *= 2;
  }

  // griddim.z should be incount
  bd_ = Dim3::fill_xyz_by_pow2(Dim3(blockLength_ / wordSize_, count_, 1), 512);
  gd_ = (Dim3(blockLength_ / wordSize_, count_, 1) + bd_ - Dim3(1, 1, 1)) / bd_;
}

void PackerStride1::launch_pack(void *outbuf, int *position, const void *inbuf,
                                const int incount, hipStream_t stream) const {
  inbuf = static_cast<const char *>(inbuf) + offset_;

  if (uintptr_t(inbuf) % wordSize_) {
    LOG_WARN("pack kernel may be unaligned.");
  }

  Dim3 gd = gd_;
  gd.z = incount;

  if (4 == wordSize_) {
    LOG_SPEW("wordSize_ = 4");
    pack_bytes<4><<<gd, bd_, 0, stream>>>(outbuf, *position, inbuf, incount,
                                          blockLength_, count_, stride_);

  } else if (8 == wordSize_) {
    LOG_SPEW("wordSize_ = 8");
    pack_bytes<8><<<gd, bd_, 0, stream>>>(outbuf, *position, inbuf, incount,
                                          blockLength_, count_, stride_);

  } else {
    LOG_SPEW("wordSize == 1");
    pack_bytes<1><<<gd, bd_, 0, stream>>>(outbuf, *position, inbuf, incount,
                                          blockLength_, count_, stride_);
  }
  CUDA_RUNTIME(hipGetLastError());
  (*position) += incount * count_ * blockLength_;
}

void PackerStride1::launch_unpack(const void *inbuf, int *position,
                                  void *outbuf, const int outcount,
                                  hipStream_t stream) const {
  outbuf = static_cast<char *>(outbuf) + offset_;

  Dim3 gd = gd_;
  gd.z = outcount;

  if (4 == wordSize_) {
    LOG_SPEW("wordSize_ = 4");
    unpack_bytes<4><<<gd, bd_, 0, stream>>>(outbuf, *position, inbuf, outcount,
                                            blockLength_, count_, stride_);

  } else if (8 == wordSize_) {
    LOG_SPEW("wordSize_ = 8");
    unpack_bytes<8><<<gd, bd_, 0, stream>>>(outbuf, *position, inbuf, outcount,
                                            blockLength_, count_, stride_);
  } else {
    LOG_SPEW("wordSize == 1");
    unpack_bytes<1><<<gd, bd_, 0, stream>>>(outbuf, *position, inbuf, outcount,
                                            blockLength_, count_, stride_);
  }
  CUDA_RUNTIME(hipGetLastError());
  (*position) += outcount * count_ * blockLength_;
}

#if 0
void PackerStride1::pack_async(void *outbuf, int *position, const void *inbuf,
                               const int incount) const {
  int device;
  CUDA_RUNTIME(hipGetDevice(&device));
  LaunchInfo info = pack_launch_info(inbuf);
  LOG_SPEW("PackerStride1::pack on CUDA " << info.device);
  CUDA_RUNTIME(hipSetDevice(info.device));
  launch_pack(outbuf, position, inbuf, incount, info.stream);
  LOG_SPEW("PackerStride1::restore device " << device);
  CUDA_RUNTIME(hipSetDevice(device));
}
#endif

// same as async but synchronize after launch
void PackerStride1::pack(void *outbuf, int *position, const void *inbuf,
                         const int incount) const {
  LaunchInfo info = pack_launch_info(inbuf);
  launch_pack(outbuf, position, inbuf, incount, info.stream);
  CUDA_RUNTIME(hipStreamSynchronize(info.stream));
}

#if 0
void PackerStride1::unpack_async(const void *inbuf, int *position, void *outbuf,
                                 const int outcount) const {
  int device;
  CUDA_RUNTIME(hipGetDevice(&device));
  LaunchInfo info = unpack_launch_info(outbuf);
  LOG_SPEW("PackerStride1::unpack on CUDA " << info.device);
  CUDA_RUNTIME(hipSetDevice(info.device));

  launch_unpack(inbuf, position, outbuf, outcount, info.stream);

  CUDA_RUNTIME(hipStreamSynchronize(info.stream));
  LOG_SPEW("PackerStride1::restore device " << device);
  CUDA_RUNTIME(hipSetDevice(device));
}
#endif

void PackerStride1::unpack(const void *inbuf, int *position, void *outbuf,
                           const int outcount) const {
  LaunchInfo info = unpack_launch_info(outbuf);
  launch_unpack(inbuf, position, outbuf, outcount, info.stream);
  CUDA_RUNTIME(hipStreamSynchronize(info.stream));
}