#include "hip/hip_runtime.h"
#include "packer_stride_1.hpp"

#include "cuda_runtime.hpp"
#include "dim3.hpp"
#include "logging.hpp"

/* pack blocks of bytes separated a stride

    the z dimension is used for the incount

each thread loads N bytes of a block
 */
template <unsigned N>
__global__ static void
pack_bytes(void *__restrict__ outbuf,
           int position, // location in the output buffer to start packing (B)
           const void *__restrict__ inbuf,
           const int incount,    // number of datatypes to pack
           unsigned blockLength, // block length (B)
           unsigned count,       // count of blocks in a group
           unsigned stride       // stride (B) between start of blocks in group
) {

  assert(blockLength % N == 0); // N should evenly divide block length

  const int extent = (count - 1) * stride + blockLength;

  const unsigned int tz = blockDim.z * blockIdx.z + threadIdx.z;
  const unsigned int ty = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned int tx = blockDim.x * blockIdx.x + threadIdx.x;

  char *__restrict__ op = reinterpret_cast<char *>(outbuf) + position;
  const char *__restrict__ ip = reinterpret_cast<const char *>(inbuf);

  for (int z = tz; z < incount; z += gridDim.z * blockDim.z) {
    // each packed datatype will take count * blockLength bytes in outbuf
    // each datatype input has stride * count separating their starts
    char *__restrict__ dst = op + z * blockLength * count;
    const char *__restrict__ src = ip + z * extent;

    // x direction handle the blocks, y handles the block counts
    for (unsigned y = ty; y < count; y += gridDim.y * blockDim.y) {
      for (unsigned x = tx; x < blockLength / N; x += gridDim.x * blockDim.x) {
        unsigned bo = y * blockLength + x * N;
        unsigned bi = y * stride + x * N;
        // printf("%u -> %u\n", bi, bo);

        if (N == 1) {
          dst[bo] = src[bi];
        } else if (N == 2) {
          uint16_t *__restrict__ d = reinterpret_cast<uint16_t *>(dst + bo);
          const uint16_t *__restrict__ s =
              reinterpret_cast<const uint16_t *>(src + bi);
          *d = *s;
        } else if (N == 4) {
          uint32_t *__restrict__ d = reinterpret_cast<uint32_t *>(dst + bo);
          const uint32_t *__restrict__ s =
              reinterpret_cast<const uint32_t *>(src + bi);
          *d = *s;
        } else if (N == 8) {
          uint64_t *__restrict__ d = reinterpret_cast<uint64_t *>(dst + bo);
          const uint64_t *__restrict__ s =
              reinterpret_cast<const uint64_t *>(src + bi);
          *d = *s;
        }
      }
    }
  }
}

/* unpack

    griddim.z is the count

each thread loads N bytes of a block
*/
template <unsigned N>
__global__ static void unpack_bytes(
    void *__restrict__ outbuf,
    int position, // location in the output buffer to start unpacking (B)
    const void *__restrict__ inbuf,
    const int outcount,   // number of datatypes to unpack
    unsigned blockLength, // block length (B)
    unsigned count,       // count of blocks in a group
    unsigned stride       // stride (B) between start of blocks in group
) {

  assert(blockLength % N == 0); // N should evenly divide block length

  const int extent = (count - 1) * stride + blockLength;

  const unsigned int tz = blockDim.z * blockIdx.z + threadIdx.z;
  const unsigned int ty = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned int tx = blockDim.x * blockIdx.x + threadIdx.x;

  char *__restrict__ op = reinterpret_cast<char *>(outbuf) + position;
  const char *__restrict__ ip = reinterpret_cast<const char *>(inbuf);

  for (int z = tz; z < outcount; z += gridDim.z * blockDim.z) {
    // each datatype will have stride * count separating their starts in outbuf
    // each packed datatype has blockLength * count separating their starts
    char *__restrict__ dst = op + z * extent;
    const char *__restrict__ src = ip + z * blockLength * count;

    // x direction handle the blocks, y handles the block counts
    for (unsigned y = ty; y < count; y += gridDim.y * blockDim.y) {
      for (unsigned x = tx; x < blockLength / N; x += gridDim.x * blockDim.x) {
        unsigned bi = y * blockLength + x * N;
        unsigned bo = y * stride + x * N;
        // printf("%u -> %u\n", bi, bo);

        if (N == 1) {
          dst[bo] = src[bi];
        } else if (N == 2) {
          uint16_t *__restrict__ d = reinterpret_cast<uint16_t *>(dst + bo);
          const uint16_t *__restrict__ s =
              reinterpret_cast<const uint16_t *>(src + bi);
          *d = *s;
        } else if (N == 4) {
          uint32_t *__restrict__ d = reinterpret_cast<uint32_t *>(dst + bo);
          const uint32_t *__restrict__ s =
              reinterpret_cast<const uint32_t *>(src + bi);
          *d = *s;
        } else if (N == 8) {
          uint64_t *__restrict__ d = reinterpret_cast<uint64_t *>(dst + bo);
          const uint64_t *__restrict__ s =
              reinterpret_cast<const uint64_t *>(src + bi);
          *d = *s;
        }
      }
    }
  }
}

PackerStride1::PackerStride1(unsigned blockLength, unsigned count,
                             unsigned stride) {
  blockLength_ = blockLength;
  count_ = count;
  stride_ = stride;

  wordSize_ = 1;
  while (0 == blockLength_ % (wordSize_ * 2) && (wordSize_ * 2 <= 4)) {
    wordSize_ *= 2;
  }

  // griddim.z should be incount
  bd_ = Dim3::fill_xyz_by_pow2(Dim3(blockLength_ / wordSize_, count_, 1), 512);
  gd_ = (Dim3(blockLength_ / wordSize_, count_, 1) + bd_ - Dim3(1, 1, 1)) / bd_;
}

void PackerStride1::pack_async(void *outbuf, int *position, const void *inbuf,
                               const int incount) const {

  int device;
  CUDA_RUNTIME(hipGetDevice(&device));
  LaunchInfo info = pack_launch_info(inbuf);
  LOG_SPEW("PackerStride1::pack on CUDA " << info.device);
  CUDA_RUNTIME(hipSetDevice(info.device));

#if 0
  char *__restrict__ op = reinterpret_cast<char *>(outbuf);
  const char *__restrict__ ip = reinterpret_cast<const char *>(inbuf);

  for (int i = 0; i < incount; ++i) {
    char *__restrict__ dst =
        op + *position + i * count_[1] * count_[0] * blockLength_;
    const char *__restrict__ src =
        ip + i * stride_[1] * count_[1] * stride_[0] * count_[0];

    for (unsigned z = 0; z < count_[1]; ++z) {
      for (unsigned y = 0; y < count_[0]; ++y) {
        for (unsigned x = 0; x < blockLength_; ++x) {
          int64_t bo = z * count_[0] * blockLength_ + y * blockLength_ + x;
          int64_t bi = z * stride_[1] + y * stride_[0] + x;
          // std::cerr << bi << " -> " << bo << "\n";
          dst[bo] = src[bi];
        }
      }
    }
  }
#endif

  Dim3 gd = gd_;
  gd.z = incount;

  if (4 == wordSize_) {
    LOG_SPEW("wordSize_ = 4");
    pack_bytes<4><<<gd, bd_, 0, info.stream>>>(
        outbuf, *position, inbuf, incount, blockLength_, count_, stride_);

  } else if (8 == wordSize_) {
    LOG_SPEW("wordSize_ = 8");
    pack_bytes<8><<<gd, bd_, 0, info.stream>>>(
        outbuf, *position, inbuf, incount, blockLength_, count_, stride_);

  } else {
    LOG_SPEW("wordSize == 1");
    pack_bytes<1><<<gd, bd_, 0, info.stream>>>(
        outbuf, *position, inbuf, incount, blockLength_, count_, stride_);
  }
  CUDA_RUNTIME(hipGetLastError());
  (*position) += incount * count_ * blockLength_;

  LOG_SPEW("PackerStride1::restore device " << device);
  CUDA_RUNTIME(hipSetDevice(device));
}

void PackerStride1::unpack(const void *inbuf, int *position, void *outbuf,
                           const int outcount) const {
  int device;
  CUDA_RUNTIME(hipGetDevice(&device));
  LaunchInfo info = unpack_launch_info(outbuf);
  LOG_SPEW("PackerStride1::unpack on CUDA " << info.device);
  CUDA_RUNTIME(hipSetDevice(info.device));

  Dim3 gd = gd_;
  gd.z = outcount;

  if (4 == wordSize_) {
    LOG_SPEW("wordSize_ = 4");
    unpack_bytes<4><<<gd, bd_, 0, info.stream>>>(
        outbuf, *position, inbuf, outcount, blockLength_, count_, stride_);

  } else if (8 == wordSize_) {
    LOG_SPEW("wordSize_ = 8");
    unpack_bytes<8><<<gd, bd_, 0, info.stream>>>(
        outbuf, *position, inbuf, outcount, blockLength_, count_, stride_);
  } else {
    LOG_SPEW("wordSize == 1");
    unpack_bytes<1><<<gd, bd_, 0, info.stream>>>(
        outbuf, *position, inbuf, outcount, blockLength_, count_, stride_);
  }

  CUDA_RUNTIME(hipGetLastError());

  (*position) += outcount * count_ * blockLength_;

  CUDA_RUNTIME(hipStreamSynchronize(info.stream));
  LOG_SPEW("PackerStride1::restore device " << device);
  CUDA_RUNTIME(hipSetDevice(device));
}