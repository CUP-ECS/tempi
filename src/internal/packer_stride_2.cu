#include "hip/hip_runtime.h"
#include "packer_stride_2.hpp"

#include "cuda_runtime.hpp"
#include "dim3.hpp"
#include "logging.hpp"
#include "streams.hpp"

/* pack blocks of bytes separated by two strides

each thread loads N bytes of a block
 */
template <unsigned N>
__global__ static void pack_bytes(
    void *__restrict__ outbuf, int position, // position in output buffer
    const void *__restrict__ inbuf, const int incount,
    unsigned blockLength, // block length (B)
    unsigned count0,      // count of inner blocks in a group
    unsigned stride0,     // stride (B) between start of inner blocks in group
    unsigned count1,      // number of block groups
    unsigned stride1      // stride (B) between start of block groups
) {

  assert(blockLength % N == 0); // N should evenly divide block length

  const unsigned int tz = blockDim.z * blockIdx.z + threadIdx.z;
  const unsigned int ty = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned int tx = blockDim.x * blockIdx.x + threadIdx.x;

  char *__restrict__ op = reinterpret_cast<char *>(outbuf) + position;
  const char *__restrict__ ip = reinterpret_cast<const char *>(inbuf);

  for (int i = 0; i < incount; ++i) {
    char *__restrict__ dst = op + i * count1 * count0 * blockLength;
    const char *__restrict__ src = ip + i * stride1 * count1 * stride0 * count0;

    for (unsigned z = tz; z < count1; z += gridDim.z * blockDim.z) {
      for (unsigned y = ty; y < count0; y += gridDim.y * blockDim.y) {
        for (unsigned x = tx; x < blockLength / N;
             x += gridDim.x * blockDim.x) {
          unsigned bo = z * count0 * blockLength + y * blockLength + x * N;
          unsigned bi = z * stride1 + y * stride0 + x * N;
          // printf("%u -> %u\n", bi, bo);

          if (N == 1) {
            dst[bo] = src[bi];
          } else if (N == 2) {
            uint16_t *__restrict__ d = reinterpret_cast<uint16_t *>(dst + bo);
            const uint16_t *__restrict__ s =
                reinterpret_cast<const uint16_t *>(src + bi);
            *d = *s;
          } else if (N == 4) {
            uint32_t *__restrict__ d = reinterpret_cast<uint32_t *>(dst + bo);
            const uint32_t *__restrict__ s =
                reinterpret_cast<const uint32_t *>(src + bi);
            *d = *s;
          } else if (N == 8) {
            uint64_t *__restrict__ d = reinterpret_cast<uint64_t *>(dst + bo);
            const uint64_t *__restrict__ s =
                reinterpret_cast<const uint64_t *>(src + bi);
            *d = *s;
          }
        }
      }
    }
  }
}

template <unsigned N>
__global__ static void unpack_bytes(
    void *__restrict__ outbuf, int position, const void *__restrict__ inbuf,
    const int incount,
    const unsigned blockLength, // block length (B)
    const unsigned count0,      // count of inner blocks in a group
    const unsigned stride0, // stride (B) between start of inner blocks in group
    const unsigned count1,  // number of block groups
    const unsigned stride1  // stride (B) between start of block groups
) {

  assert(blockLength % N == 0); // N should evenly divide block length

  const unsigned int tz = blockDim.z * blockIdx.z + threadIdx.z;
  const unsigned int ty = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned int tx = blockDim.x * blockIdx.x + threadIdx.x;

  char *__restrict__ op = reinterpret_cast<char *>(outbuf) + position;
  const char *__restrict__ ip = reinterpret_cast<const char *>(inbuf);

  for (int i = 0; i < incount; ++i) {
    char *__restrict__ dst = op + i * stride1 * count1 * stride0 * count0;
    const char *__restrict__ src = ip + i * count1 * count0 * blockLength;

    for (unsigned z = tz; z < count1; z += gridDim.z * blockDim.z) {
      for (unsigned y = ty; y < count0; y += gridDim.y * blockDim.y) {
        for (unsigned x = tx; x < blockLength / N;
             x += gridDim.x * blockDim.x) {
          unsigned bi = z * count0 * blockLength + y * blockLength + x * N;
          unsigned bo = z * stride1 + y * stride0 + x * N;
          // printf("%u -> %u\n", bi, bo);

          if (N == 1) {
            dst[bo] = src[bi];
          } else if (N == 2) {
            uint16_t *__restrict__ d = reinterpret_cast<uint16_t *>(dst + bo);
            const uint16_t *__restrict__ s =
                reinterpret_cast<const uint16_t *>(src + bi);
            *d = *s;
          } else if (N == 4) {
            uint32_t *__restrict__ d = reinterpret_cast<uint32_t *>(dst + bo);
            const uint32_t *__restrict__ s =
                reinterpret_cast<const uint32_t *>(src + bi);
            *d = *s;
          } else if (N == 8) {
            uint64_t *__restrict__ d = reinterpret_cast<uint64_t *>(dst + bo);
            const uint64_t *__restrict__ s =
                reinterpret_cast<const uint64_t *>(src + bi);
            *d = *s;
          }
        }
      }
    }
  }
}

PackerStride2::PackerStride2(unsigned blockLength, unsigned count0,
                             unsigned stride0, unsigned count1,
                             unsigned stride1) {
  blockLength_ = blockLength;
  count_[0] = count0;
  count_[1] = count1;
  stride_[0] = stride0;
  stride_[1] = stride1;

  wordSize_ = 1;
  while (0 == blockLength % (wordSize_ * 2) && (wordSize_ * 2 <= 4)) {
    wordSize_ *= 2;
  }

  bd_ = Dim3::fill_xyz_by_pow2(
      Dim3(blockLength_ / wordSize_, count_[0], count_[1]), 512);
  gd_ = (Dim3(blockLength_ / wordSize_, count_[0], count_[1]) + bd_ -
         Dim3(1, 1, 1)) /
        bd_;
}

void PackerStride2::pack(void *outbuf, int *position, const void *inbuf,
                         const int incount) const {

  int device;
  CUDA_RUNTIME(hipGetDevice(&device));
  LOG_SPEW("PackerStride2::pack() on CUDA " << device);

  assert(kernStream.size() > 0 && "no streams. Are GPUs enabled and was MPI_Init called?");
  hipStream_t stream = kernStream[device];


#if 0
  char *__restrict__ op = reinterpret_cast<char *>(outbuf);
  const char *__restrict__ ip = reinterpret_cast<const char *>(inbuf);

  for (int i = 0; i < incount; ++i) {
    char *__restrict__ dst =
        op + *position + i * count_[1] * count_[0] * blockLength_;
    const char *__restrict__ src =
        ip + i * stride_[1] * count_[1] * stride_[0] * count_[0];

    for (unsigned z = 0; z < count_[1]; ++z) {
      for (unsigned y = 0; y < count_[0]; ++y) {
        for (unsigned x = 0; x < blockLength_; ++x) {
          int64_t bo = z * count_[0] * blockLength_ + y * blockLength_ + x;
          int64_t bi = z * stride_[1] + y * stride_[0] + x;
          // std::cerr << bi << " -> " << bo << "\n";
          dst[bo] = src[bi];
        }
      }
    }
  }
#endif

  if (4 == wordSize_) {
    LOG_SPEW("wordSize_ = 4");
    pack_bytes<4><<<gd_, bd_, 0, kernStream[device]>>>(
        outbuf, *position, inbuf, incount, blockLength_, count_[0], stride_[0],
        count_[1], stride_[1]);

  } else if (8 == wordSize_) {
    LOG_SPEW("wordSize_ = 8");
    pack_bytes<8><<<gd_, bd_, 0, kernStream[device]>>>(
        outbuf, *position, inbuf, incount, blockLength_, count_[0], stride_[0],
        count_[1], stride_[1]);

  } else {
    LOG_SPEW("wordSize == 1");
    pack_bytes<1><<<gd_, bd_, 0, kernStream[device]>>>(
        outbuf, *position, inbuf, incount, blockLength_, count_[0], stride_[0],
        count_[1], stride_[1]);
  }

  CUDA_RUNTIME(hipGetLastError());

  assert(position);
  (*position) += incount * count_[1] * count_[0] * blockLength_;

  CUDA_RUNTIME(hipStreamSynchronize(kernStream[device]));
}

void PackerStride2::unpack(const void *inbuf, int *position, void *outbuf,
                           const int outcount) const {

  int device;
  CUDA_RUNTIME(hipGetDevice(&device));
  LOG_SPEW("PackerStride2::unpack() on CUDA " << device);

  if (4 == wordSize_) {
    LOG_SPEW("wordSize_ = 4");
    unpack_bytes<4><<<gd_, bd_, 0, kernStream[device]>>>(
        outbuf, *position, inbuf, outcount, blockLength_, count_[0], stride_[0],
        count_[1], stride_[1]);

  } else if (8 == wordSize_) {
    LOG_SPEW("wordSize_ = 8");
    unpack_bytes<8><<<gd_, bd_, 0, kernStream[device]>>>(
        outbuf, *position, inbuf, outcount, blockLength_, count_[0], stride_[0],
        count_[1], stride_[1]);

  } else {
    LOG_SPEW("wordSize == 1");
    unpack_bytes<1><<<gd_, bd_, 0, kernStream[device]>>>(
        outbuf, *position, inbuf, outcount, blockLength_, count_[0], stride_[0],
        count_[1], stride_[1]);
  }

  CUDA_RUNTIME(hipGetLastError());

  (*position) += outcount * count_[1] * count_[0] * blockLength_;

  CUDA_RUNTIME(hipStreamSynchronize(kernStream[device]));
}

