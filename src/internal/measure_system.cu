#include "hip/hip_runtime.h"
#include "benchmark.hpp"
#include "cuda_runtime.hpp"
#include "measure_system.hpp"

#include <mpi.h>

#include <chrono>
#include <iostream>

typedef std::chrono::high_resolution_clock Clock;
typedef std::chrono::duration<double> Duration;
typedef std::chrono::time_point<Clock, Duration> Time;

/* extern*/ KernelLaunch kernelLaunch;

static __global__ void kernel(int *a) {
  if (a) {
    *a = threadIdx.x;
  }
}

class KernelLaunchBenchmark : public Benchmark {
  hipStream_t stream;

public:
  KernelLaunchBenchmark() { CUDA_RUNTIME(hipStreamCreate(&stream)); }

  ~KernelLaunchBenchmark() { CUDA_RUNTIME(hipStreamDestroy(stream)); }

  Benchmark::IterResult run_iter() override {
    IterResult res{};

    Time start = Clock::now();
    for (int i = 0; i < 32; ++i) {
      kernel<<<1, 1, 0, stream>>>(nullptr);
    }
    Time stop = Clock::now();
    Duration dur = stop - start;
    CUDA_RUNTIME(hipStreamSynchronize(stream));

    res.time = dur.count() / 32.0;
    return res;
  }
};

void measure_system(MPI_Comm comm) {

  int rank, size;
  MPI_Comm_rank(comm, &rank);
  MPI_Comm_size(comm, &size);

  if (rank == 0) {
    Benchmark *bm = new KernelLaunchBenchmark();
    Benchmark::Result res = bm->run();
    std::cerr << "=== " << res.trimean << " " << res.nIters << " ===\n";
    delete bm;
    kernelLaunch.secs = res.trimean;
  }
}
