//          Copyright Carl Pearson 2020 - 2021.
// Distributed under the Boost Software License, Version 1.0.
//    (See accompanying file LICENSE or copy at
//    https://www.boost.org/LICENSE_1_0.txt)

#include "pack_kernels.cuh"
#include "packer_2d.hpp"

#include "counters.hpp"
#include "cuda_runtime.hpp"
#include "dim3.hpp"
#include "logging.hpp"

Packer2D::Packer2D(unsigned off, unsigned blockLength, unsigned count,
                   unsigned stride, unsigned extent)
    : offset_(off), blockLength_(blockLength), count_(count), stride_(stride),
      extent_(extent), config_(off, blockLength, count) {
  assert(blockLength_ > 0);

}

void Packer2D::launch_pack(void *outbuf, int *position, const void *inbuf,
                           const int incount, hipStream_t stream,
                           hipEvent_t kernelStart,
                           hipEvent_t kernelStop) const {
  TEMPI_COUNTER_OP(pack2d, NUM_PACKS, ++);
  inbuf = static_cast<const char *>(inbuf) + offset_;

  const dim3 gd = config_.dim_grid(incount);
  const dim3 bd = config_.dim_block();
  if (kernelStart) {
    CUDA_RUNTIME(hipEventRecord(kernelStart, stream));
  }
  outbuf = static_cast<char *>(outbuf) + *position;
  LOG_SPEW("packfn_");
  config_.packfn<<<gd, bd, 0, stream>>>(outbuf, inbuf, incount, blockLength_, count_,
                                  stride_, extent_);
  if (kernelStop) {
    CUDA_RUNTIME(hipEventRecord(kernelStop, stream));
  }
  CUDA_RUNTIME(hipGetLastError());
  (*position) += incount * count_ * blockLength_;
}

void Packer2D::launch_unpack(const void *inbuf, int *position, void *outbuf,
                             const int outcount, hipStream_t stream,
                             hipEvent_t kernelStart,
                             hipEvent_t kernelStop) const {
  TEMPI_COUNTER_OP(pack2d, NUM_UNPACKS, ++);
  outbuf = static_cast<char *>(outbuf) + offset_;

  const dim3 gd = config_.dim_grid(outcount);
  const dim3 bd = config_.dim_block();

  if (kernelStart) {
    CUDA_RUNTIME(hipEventRecord(kernelStart, stream));
  }

  outbuf = static_cast<char *>(outbuf) + *position;
  config_.unpackfn<<<gd, bd, 0, stream>>>(outbuf, inbuf, outcount, blockLength_,
                                    count_, stride_, extent_);

  if (kernelStop) {
    CUDA_RUNTIME(hipEventRecord(kernelStop, stream));
  }
  CUDA_RUNTIME(hipGetLastError());
  (*position) += outcount * count_ * blockLength_;
}

void Packer2D::pack_async(void *outbuf, int *position, const void *inbuf,
                          const int incount, hipEvent_t event) const {
  LaunchInfo info = pack_launch_info(inbuf);
  launch_pack(outbuf, position, inbuf, incount, info.stream);
  if (event) {
    CUDA_RUNTIME(hipEventRecord(event, info.stream));
  }
}

void Packer2D::unpack_async(const void *inbuf, int *position, void *outbuf,
                            const int outcount, hipEvent_t event) const {
  LaunchInfo info = unpack_launch_info(outbuf);
  launch_unpack(inbuf, position, outbuf, outcount, info.stream);
  if (event) {
    CUDA_RUNTIME(hipEventRecord(event, info.stream));
  }
}

// same as async but synchronize after launch
void Packer2D::pack(void *outbuf, int *position, const void *inbuf,
                    const int incount) const {
  LaunchInfo info = pack_launch_info(inbuf);
  launch_pack(outbuf, position, inbuf, incount, info.stream);
  CUDA_RUNTIME(hipStreamSynchronize(info.stream));
}

void Packer2D::unpack(const void *inbuf, int *position, void *outbuf,
                      const int outcount) const {
  LaunchInfo info = unpack_launch_info(outbuf);
  launch_unpack(inbuf, position, outbuf, outcount, info.stream);
  CUDA_RUNTIME(hipStreamSynchronize(info.stream));
}